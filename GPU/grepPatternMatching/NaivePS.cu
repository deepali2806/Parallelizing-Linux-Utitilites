#include<stdio.h>
#include<stdlib.h>
#include <sys/time.h> 
#include <climits>
#include <fstream>
#include <iostream>
#include <sstream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand.h>
#include<bits/stdc++.h>
#define BLOCKSIZE 32

using namespace std;

__global__ void patternMatchingNaive(char *gpuInputText ,int n, int length, char *pattern, int* gpuResultLine, int pLength, int *gpuLength)
{
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < n)
    {
       // printf("\nIn pattern match %c", gpuInputText[id*length + 0]);
      //  printf("\n Pattern is %s", pattern );
        //Considering reusltMatrix has fix length for all lines       
        int i = 0;
        while(i < (gpuLength[id] - 1))
        {
           // printf("\nIn For loop pattern match");
            int cnt = 0, j = 0;
            int temp = i;
            while(pattern[j] != '\0')
            {
                if(temp < length && pattern[j] == gpuInputText[id*length + temp])   
                {
                    cnt++;
                }
                else
                {
                    break;
                }
                j++;
                temp++;
            }
            
           // printf("Count %d", cnt);

            if(cnt == pLength)
            { 
                gpuResultLine[id] = 1;
             //   printf("\n Found ");

                // for(int k = 0; k < length; k++)
                // {
                //     if(gpuInputText[id*length + k] != '\0')
                //         printf("%d %c", id, gpuInputText[id*length + k]);
                //     else 
                //         break;
                // }
                    
               // printf("\n");
                return;
           }
           
           i++;
        }
        gpuResultLine[id] = 0;

    }
}


int main(int argc, char **argv)
{
    int n;
    int iLength;
   // n = 8;
    
	FILE *filePointer;
	char *filename = argv[1]; 
   	filePointer = fopen( filename , "r") ; 
      
    if ( filePointer == NULL ) 
    {
       printf( "input.txt file failed to open." ) ; 
	   return 0;
    }
    char newLine;
    fscanf(filePointer, "%d", &n);
    fscanf(filePointer, "%d", &iLength);
    fscanf(filePointer, "%c", &newLine);

    char inputText[(iLength+1)*(n)];
    int length[n];
    char temp;

    //Todo last line may not havenewLine character at end
    for (int i = 0; i < n; i++)
    {
        int temp1 = 0;
        for(int j = 0; j <= iLength; j++)
        {
            temp1++;
            fscanf(filePointer, "%c", &temp);
            if(temp == '\n')
            {
                inputText[i*iLength + j] = '\0';
                break;
            }  
            else
            {
                inputText[i*iLength + j] = temp;
    
            }
        }
        length[i] = temp1;
    }
  
    // cout<<"Length";
    // for(int i = 0; i < n; i++)
    // {
    //     cout<<length[i]<<" ";
    // }
    // cout<<endl;
    // for (size_t i = 0; i < n; i++)
	// {
	//     for (size_t j = 0; j <= iLength; j++)
	//     {
	//     	if(inputText[i*iLength + j] != '\0')
	// 		printf("%c", inputText[i*iLength + j]);
    //         else
    //         {
    //             printf("Null");
    //             break;
    //         }
	//     }
	//     printf("\n");
	// }

    char *gpuInputText;
    hipMalloc(&gpuInputText, (iLength+1)*(n)*sizeof(char));
    hipMemcpy(gpuInputText, inputText, (iLength+1)*(n)*sizeof(char), hipMemcpyHostToDevice);

    int *gpuLength;
    hipMalloc( &gpuLength, sizeof(int)*n);
    hipMemcpy(gpuLength, length,  sizeof(int)*n, hipMemcpyHostToDevice);

    char pattern[20];
    printf("Enter the pattern you want to find");
    scanf("%s", pattern);
    printf("\nPattern is %s\n", pattern);
    
    int pLength = 0;
    while (pattern[pLength] != '\0')
    {
        pLength++;
    }
    

    char *gpuPattern;
    hipMalloc(&gpuPattern, 20);
    hipMemcpy(gpuPattern, pattern, 20 , hipMemcpyHostToDevice);

    int *results = (int *)malloc(n*sizeof(int));
	memset(results, 0, n*sizeof(int));

    int *gpuResultLine;
    hipMalloc( &gpuResultLine, sizeof(int)*n);
    hipMemcpy(gpuResultLine, results,  sizeof(int)*n, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    hipEventRecord(start,0);


    patternMatchingNaive <<<ceil(float(n)/BLOCKSIZE) ,BLOCKSIZE>>> (gpuInputText, n,iLength, gpuPattern, gpuResultLine, pLength, gpuLength);
    hipDeviceSynchronize();

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken by function to execute is: %.6f ms\n", milliseconds);
    

    hipMemcpy(results, gpuResultLine, sizeof(int)*n , hipMemcpyDeviceToHost);


    char *fname = argv[2]; 
    FILE *fptr;
    fptr = fopen(fname,"w");
    
    //cout<<"Final";

    
    // for(int i=0;i<n;i++)
    // {
    //     cout<<results[i]<<" ";
    // }

   // cout<<endl;
    for(int i=0;i<=n;i++)
    {
        if(results[i] == 1)
        {
            for(int j = 0; j < iLength; j++)
            {
                if(inputText[i*iLength + j] != '\0')
                    fprintf(fptr ,"%c", inputText[i*iLength + j] ); 
                else
                    break;
            }
            fprintf(fptr,"\n");
        }
    }

    fprintf(fptr,"\n"); 



    return 0;
}