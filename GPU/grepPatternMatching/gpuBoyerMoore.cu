#include<stdio.h>
#include<stdlib.h>
#include <sys/time.h> 
#include <climits>
#include <fstream>
#include <iostream>
#include <sstream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand.h>
#include<bits/stdc++.h>
#define BLOCKSIZE 32

using namespace std;


__global__ void patternMatchingBoyerMoore(char *inputText, int iLength, int *gpuLength, int *lastTable, char *pattern, int pLength, int* gpuResultLine, int n)
{
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < n)
    {
       // printf("\nId  %d Inside Last Table\n", id);
        
        if(gpuLength[id] < pLength)
        {
            gpuResultLine[id] = 0;
            return;
        }

        int i = pLength - 1;
        int j= pLength - 1;
        int cnt = 0;
        while(i < (gpuLength[id] - 1))
        {

           // printf("\nId %d inpt Char %c iLength %d i %d", id, inputText[id*iLength + i], iLength, i);
            if(inputText[id*iLength + i] == pattern[j])
            {
                if(j == 0)
                {
                    //printf("Found Id %d", id);
                    gpuResultLine[id] = 1;
                    return;
                }
                else
                {
                    i--;
                    j--;
                }
            }
            else
            {
                if(inputText[id*iLength + i] > 57)
                {
                    if(inputText[id*iLength + i] >= 97 && inputText[id*iLength + i] <= 122)
                    {
                        i = i + pLength - min (j, 1 + lastTable[inputText[id*iLength + i] - 97 + 26]);
                    }
                    else
                    {
          //              printf("\nId %d j %d lastTabe %d inpt Char %c", id, j, lastTable[inputText[id*iLength + i] - 65], inputText[id*iLength + i]);
                        i = i + pLength - min (j, 1 + lastTable[inputText[id*iLength + i] - 65]);
                    }
                }
                else
                {
                    if(inputText[id*iLength + i] >= 48 && inputText[id*iLength + i] <= 57)
                        i = i + pLength - min (j, 1 + lastTable[inputText[id*iLength + i] - 48 + 52]);
                    else if(inputText[id*iLength + i] == 32)
                        i = i + pLength - min (j, 1 + lastTable[inputText[id*iLength + i] - 32 + 62]);
                }

                j = pLength - 1;
            }

         //   printf("\nId %d Shift %d", id, i);
         cnt++;
        }

        gpuResultLine[id] = 0;
        return;
       
    } 
}



int main(int argc, char **argv)
{
    int n;
    int iLength;
    
	FILE *filePointer;
	char *filename = argv[1]; 
   	filePointer = fopen( filename , "r") ; 
      
    if ( filePointer == NULL ) 
    {
       printf( "input.txt file failed to open." ) ; 
	   return 0;
    }

    char newLine;
    fscanf(filePointer, "%d", &n);
    fscanf(filePointer, "%d", &iLength);
    fscanf(filePointer, "%c", &newLine);

    char inputText[(iLength+1)*(n)];
    int length[n];
    char temp;

    //Todo last line may not havenewLine character at end
    for (int i = 0; i < n; i++)
    {
        int temp1 = 0;
        for(int j = 0; j <= iLength; j++)
        {
            temp1++;
            fscanf(filePointer, "%c", &temp);
            if(temp == '\n')
            {
                inputText[i*iLength + j] = '\0';
                break;
            }  
            else
            {
                inputText[i*iLength + j] = temp;
    
            }
        }
        length[i] = temp1;
    }
  
    // cout<<"Length";
    // for(int i = 0; i < n; i++)
    // {
    //     cout<<length[i]<<" ";
    // }
    // cout<<endl;
    // for (size_t i = 0; i < n; i++)
	// {
	//     for (size_t j = 0; j <= iLength; j++)
	//     {
	//     	if(inputText[i*iLength + j] != '\0')
	// 		printf("%c", inputText[i*iLength + j]);
    //         else
    //         {
    //             printf("Null");
    //             break;
    //         }
	//     }
	//     printf("\n");
	// }

    char *gpuInputText;
    hipMalloc(&gpuInputText, (iLength+1)*(n)*sizeof(char));
    hipMemcpy(gpuInputText, inputText, (iLength+1)*(n)*sizeof(char), hipMemcpyHostToDevice);

    char pattern[20];
    printf("Enter the pattern you want to find");
    cin.getline(pattern, 20);
    //scanf("%s", pattern);
    printf("\nPattern is %s\n", pattern);
    
    int pLength = 0;
    while (pattern[pLength] != '\0')
    {
        pLength++;
    }
      cout<<"\nPlength "<<pLength<<endl;

    
    int *results = (int *)malloc(n*sizeof(int));
	memset(results, 0, n*sizeof(int));

    int *gpuResultLine;
    hipMalloc( &gpuResultLine, sizeof(int)*n);
    hipMemcpy(gpuResultLine, results,  sizeof(int)*n, hipMemcpyHostToDevice);

    int *gpuLength;
    hipMalloc( &gpuLength, sizeof(int)*n);
    hipMemcpy(gpuLength, length,  sizeof(int)*n, hipMemcpyHostToDevice);
    
    char *gpuPattern;
    hipMalloc(&gpuPattern, 20*sizeof(char));
    hipMemcpy(gpuPattern, pattern, 20*sizeof(char) , hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    hipEventRecord(start,0);
  
    
    //creating last Character Table : finds last occurence
    //A-Za-z0-9BlankSpace
    int lastTableSize = 63;
    int lastTable[lastTableSize];
    std::fill_n(lastTable, lastTableSize, -1);
    
    for (int i = pLength - 1; i >= 0; i--)
    {
        if (isalpha(pattern[i]))
        {
            if(islower(pattern[i]))
            {
                if(lastTable[pattern[i] - 97 + 26] == -1)
                    lastTable[pattern[i] - 97 + 26] = i;
            }
            else
            {
                if(lastTable[pattern[i] - 65] == -1)
                    lastTable[pattern[i] - 65] = i;
            }
        }
        else
        {
            if(lastTable[pattern[i] - 48 + 52] == -1)
                    lastTable[pattern[i] - 48 + 52] = i;
            else if(lastTable[pattern[i] - 32 + 62] == -1)
                    lastTable[pattern[i] - 32 + 62] = i;
        }
        
        
    }

  
   
    int *gpuLastTable;
    hipMalloc(&gpuLastTable, lastTableSize*sizeof(int));
    hipMemcpy(gpuLastTable, lastTable, lastTableSize*sizeof(int) , hipMemcpyHostToDevice);

    
    patternMatchingBoyerMoore <<<ceil(float(n)/BLOCKSIZE) ,BLOCKSIZE>>> (gpuInputText, iLength, gpuLength,gpuLastTable, gpuPattern, pLength, gpuResultLine, n);
    hipDeviceSynchronize();

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken by function to execute is: %.6f ms\n", milliseconds);
   
    hipMemcpy(results, gpuResultLine, sizeof(int)*n , hipMemcpyDeviceToHost);

    char *fname = argv[2]; 
    FILE *fptr;
    fptr = fopen(fname,"w");
    
    // cout<<"Final";

    
    // for(int i=0;i<n;i++)
    // {
    //     cout<<results[i]<<" ";
    // }

    // cout<<endl;

    for(int i=0;i<n;i++)
    {
        if(results[i] == 1)
        {
            for(int j = 0; j <= iLength; j++)
            {
                if(inputText[i*iLength + j] != '\0')
                    fprintf(fptr ,"%c", inputText[i*iLength + j] ); 
                else
                    break;
            }
            fprintf(fptr,"\n");
        }
    }
    fprintf(fptr,"\n"); 

    return 0;
}