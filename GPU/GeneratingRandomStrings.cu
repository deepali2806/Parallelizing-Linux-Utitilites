#include<stdio.h>
#include<stdlib.h>
#include <sys/time.h> 
#include <climits>
#include <fstream>
#include <iostream>
#include <sstream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand.h>
#include<bits/stdc++.h>
#define BLOCKSIZE 32

using namespace std;

__global__ void kernelForLoop(int n, int length, char *gpuResultMatrix, char *gpuCharacters, int size, hiprandState *states) 
{
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(id, id, 0, &states[id]);  // 	Initialize CURAND
   // printf("\nId %d", id);
    if(id < n)
    {
        for(int i =0; i<length; i++)
        {
                float x = hiprand_uniform (&states[id])*1000000;
                int modValue = int(x)%(size-1);
                gpuResultMatrix[id*length + i] = gpuCharacters[modValue];
            // printf(" gpuCharacters[modValue]: %c modval %d\n",gpuCharacters[modValue], modValue);
        }
    }
}


int main(int argc, char **argv)
{
    char *fname = argv[1]; 
    FILE *fptr;
    fptr = fopen(fname,"w");

    int n, length;
    printf("Enter how many lines you want to print\n");
    scanf("%d", &n);
    printf("Enter length of each string\n");
    scanf("%d", &length);
    fprintf(fptr ,"%d ", n); 
    fprintf(fptr ,"%d\n", length); 

    //n = 3;
    //length = 7;
    char *gpuResultMatrix;
    hipMalloc( &gpuResultMatrix, sizeof(char)*n*length);
    
    char characters[] = "ABCDEFGHIJKLMNOPQRSTUVWXYZabcdefghijklmnopqrstuvwxyz0123456789 ";
   //char characters[] = "ABCDEFGHIJKLMNOPQRSTUVWXYZ"; 
   int size = sizeof(characters);

    cout<<"Size :"<<size<<endl;

    char *gpuCharacters;
    hipMalloc(&gpuCharacters, size);
    hipMemcpy(gpuCharacters, characters, size , hipMemcpyHostToDevice);

    //Cuda Random states
    hiprandState *dev_random;
    hipMalloc((void**)&dev_random, (float(n)/BLOCKSIZE)*BLOCKSIZE*sizeof(hiprandState));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    hipEventRecord(start,0);


    kernelForLoop<<<ceil(float(n)/BLOCKSIZE) ,BLOCKSIZE>>> ( n, length, gpuResultMatrix, gpuCharacters, size, dev_random);
    hipDeviceSynchronize();


    char *results = (char *)malloc(n*length*sizeof(char));
    hipMemcpy(results, gpuResultMatrix, sizeof(char)*n*length , hipMemcpyDeviceToHost);

         
    for(int i=0;i<n;i++)
    {
        for(int j = 0; j < length; j++)
            fprintf(fptr ,"%c", results[i*length + j] ); 
        fprintf(fptr,"\n"); 
    }
   // fprintf(fptr,"\n"); 

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken by function to execute is: %.6f ms\n", milliseconds);
    

    return 0;
}