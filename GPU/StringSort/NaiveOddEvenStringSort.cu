#include<stdio.h>
#include<stdlib.h>
#include <sys/time.h> 
#include <climits>
#include <fstream>
#include <iostream>
#include <sstream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand.h>
#include<bits/stdc++.h>
#define BLOCKSIZE 32

using namespace std;

__global__ void naiveStringSort(char *gpuInputText, int i, int n, int *gpuIndex, int iLength, int *gpuLength)
{
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < n/2)
	{
		//Even Phase
		if((i%2 == 0) && ((id*2+1)< n))
		{
           // printf("\nEven Phase");
            //String comparing
            int flag = 0, s1 = gpuIndex[id*2], s2 = gpuIndex[id*2 + 1];
            //printf("S1 %d S2 %d", s1, s2);
            int m = gpuLength[s1];
            int n = gpuLength[s2];
           // printf("\nM %d and N %d", m, n);
            if(m == n)
            {
               // printf("\nM == n");

                for(int i = 0; i < m; i++)
                {
                 //   printf("\nId %d %c %c", id, gpuInputText[s1*iLength + i], gpuInputText[s2*iLength + i]);

                    if(gpuInputText[s1*iLength + i] <= 122 && 97 <= gpuInputText[s1*iLength + i] )
                        {
                            gpuInputText[s1*iLength + i] = gpuInputText[s1*iLength + i]- 97 + 65;
                        }
                    if(gpuInputText[s2*iLength + i] <= 122 && 97 <= gpuInputText[s2*iLength + i] )
                        {
                            gpuInputText[s2*iLength + i] = gpuInputText[s2*iLength + i]- 97 + 65;
                        }
    
                 //   printf("\nBefore Comparing %c and %c flag %d", gpuInputText[s1*iLength + i], gpuInputText[s2*iLength + i], flag);
                    if(gpuInputText[s1*iLength + i] != gpuInputText[s2*iLength + i])
                    {
                        if(gpuInputText[s1*iLength + i] > gpuInputText[s2*iLength + i])
                        {
                            flag = 1;
                            //printf("\nInside inequality Comparing %c and %c flag %d", gpuInputText[s1*iLength + i], gpuInputText[s2*iLength + i], flag);
    
                            break;   
                        }
                        else
                        {
                            //printf("\nInside inequality else Comparing %c and %c flag %d", gpuInputText[s1*iLength + i], gpuInputText[s2*iLength + i], flag);
                            break;
                        }
                    }
                   // printf("\nAfter Comparing %c and %c flag %d", gpuInputText[s1*iLength + i], gpuInputText[s2*iLength + i], flag);
    
                }
            }
            else if(m > n)
            {
                int flag1 = 0;
              //  printf("\nM > n");
                for(int i = 0; i < n; i++)
                {
               //     printf("\nId %d %c %c", id, gpuInputText[s1*iLength + i], gpuInputText[s2*iLength + i]);

                    if(gpuInputText[s1*iLength + i] <= 122 && 97 <= gpuInputText[s1*iLength + i] )
                        {
                            gpuInputText[s1*iLength + i] = gpuInputText[s1*iLength + i]- 97 + 65;
                        }
                        if(gpuInputText[s2*iLength + i] <= 122 && 97 <= gpuInputText[s2*iLength + i] )
                        {
                            gpuInputText[s2*iLength + i] = gpuInputText[s2*iLength + i]- 97 + 65;
                        }
    
               //     printf("\nBefore Comparing %c and %c flag %d", gpuInputText[s1*iLength + i], gpuInputText[s2*iLength + i], flag);
                    if(gpuInputText[s1*iLength + i] != gpuInputText[s2*iLength + i])
                    {
                        if(gpuInputText[s1*iLength + i] > gpuInputText[s2*iLength + i])
                        {
                            flag = 1;
                 //           printf("\nInside inequality Comparing %c and %c flag %d", gpuInputText[s1*iLength + i], gpuInputText[s2*iLength + i], flag);
    
                            break;   
                        }
                        else
                        {
                            flag1 = 1;
                 //           printf("\nInside inequality else Comparing %c and %c flag %d", gpuInputText[s1*iLength + i], gpuInputText[s2*iLength + i], flag);
                            break;
                        }
                    }
                   //printf("\nAfter Comparing %c and %c flag %d", gpuInputText[s1*iLength + i], gpuInputText[s2*iLength + i], flag);
    
                }

                if(flag1 == 0)
                {
                    flag = 1;
                }

            }
            else if(m < n)
            {
                //printf("\nM < n");

                for(int i = 0; i < m; i++)
                {
                 //   printf("\nId %d %c %c", id, gpuInputText[s1*iLength + i], gpuInputText[s2*iLength + i]);

                    if(gpuInputText[s1*iLength + i] <= 122 && 97 <= gpuInputText[s1*iLength + i] )
                        {
                            gpuInputText[s1*iLength + i] = gpuInputText[s1*iLength + i]- 97 + 65;
                        }
                        if(gpuInputText[s2*iLength + i] <= 122 && 97 <= gpuInputText[s2*iLength + i] )
                        {
                            gpuInputText[s2*iLength + i] = gpuInputText[s2*iLength + i]- 97 + 65;
                        }
    
                 //   printf("\nBefore Comparing %c and %c flag %d", gpuInputText[s1*iLength + i], gpuInputText[s2*iLength + i], flag);
                    if(gpuInputText[s1*iLength + i] != gpuInputText[s2*iLength + i])
                    {
                        if(gpuInputText[s1*iLength + i] > gpuInputText[s2*iLength + i])
                        {
                            flag = 1;
                  //          printf("\nInside inequality Comparing %c and %c flag %d", gpuInputText[s1*iLength + i], gpuInputText[s2*iLength + i], flag);
    
                            break;   
                        }
                        else
                        {
                   //         printf("\nInside inequality else Comparing %c and %c flag %d", gpuInputText[s1*iLength + i], gpuInputText[s2*iLength + i], flag);
                            break;
                        }
                    }
                   // printf("\nAfter Comparing %c and %c flag %d", gpuInputText[s1*iLength + i], gpuInputText[s2*iLength + i], flag);
    
                }
            }


           


			if(flag == 1)
			{
               // printf("\nSwapping");
				int temp=gpuIndex[id*2];
				gpuIndex[id*2]=gpuIndex[id*2+1];
				gpuIndex[id*2+1]=temp;	
                // for(int i =0; i < n; i++)
                //     printf("%d ",gpuIndex[i]);			
			}
		}
		
		//Odd Phase
		if((i%2 == 1) && ((id*2+2)< n))
		{
            //printf("\nOdd Phase");

            int flag = 0, s1 = gpuIndex[id*2+1], s2 = gpuIndex[id*2 + 2];
           // printf("S1 %d S2 %d", s1, s2);
           int m = gpuLength[s1];
           int n = gpuLength[s2];
          // printf("\nM %d and N %d", m, n);

           if(m == n)
           {
           // printf("\nM == n");

               for(int i = 0; i < m; i++)
               {
             //   printf("\nId %d %c %c", id, gpuInputText[s1*iLength + i], gpuInputText[s2*iLength + i]);

                   if(gpuInputText[s1*iLength + i] <= 122 && 97 <= gpuInputText[s1*iLength + i] )
                       {
                           gpuInputText[s1*iLength + i] = gpuInputText[s1*iLength + i]- 97 + 65;
                       }
                       if(gpuInputText[s2*iLength + i] <= 122 && 97 <= gpuInputText[s2*iLength + i] )
                        {
                            gpuInputText[s2*iLength + i] = gpuInputText[s2*iLength + i]- 97 + 65;
                        }
   
               //    printf("\nBefore Comparing %c and %c flag %d", gpuInputText[s1*iLength + i], gpuInputText[s2*iLength + i], flag);
                   if(gpuInputText[s1*iLength + i] != gpuInputText[s2*iLength + i])
                   {
                       if(gpuInputText[s1*iLength + i] > gpuInputText[s2*iLength + i])
                       {
                           flag = 1;
                 //          printf("\nInside inequality Comparing %c and %c flag %d", gpuInputText[s1*iLength + i], gpuInputText[s2*iLength + i], flag);
   
                           break;   
                       }
                       else
                       {
                  //         printf("\nInside inequality else Comparing %c and %c flag %d", gpuInputText[s1*iLength + i], gpuInputText[s2*iLength + i], flag);
                           break;
                       }
                   }
                //   printf("\nAfter Comparing %c and %c flag %d", gpuInputText[s1*iLength + i], gpuInputText[s2*iLength + i], flag);
   
               }
           }
           else if(m > n)
           {
               int flag1 = 0;
          //  printf("\nM > n");

               for(int i = 0; i < n; i++)
               {
          //      printf("\nId %d %c %c", id, gpuInputText[s1*iLength + i], gpuInputText[s2*iLength + i]);

                   if(gpuInputText[s1*iLength + i] <= 122 && 97 <= gpuInputText[s1*iLength + i] )
                       {
                           gpuInputText[s1*iLength + i] = gpuInputText[s1*iLength + i]- 97 + 65;
                       }
                       if(gpuInputText[s2*iLength + i] <= 122 && 97 <= gpuInputText[s2*iLength + i] )
                        {
                            gpuInputText[s2*iLength + i] = gpuInputText[s2*iLength + i]- 97 + 65;
                        }
   
            //       printf("\nBefore Comparing %c and %c flag %d", gpuInputText[s1*iLength + i], gpuInputText[s2*iLength + i], flag);
                   if(gpuInputText[s1*iLength + i] != gpuInputText[s2*iLength + i])
                   {
                       if(gpuInputText[s1*iLength + i] > gpuInputText[s2*iLength + i])
                       {
                           flag = 1;
              //             printf("\nInside inequality Comparing %c and %c flag %d", gpuInputText[s1*iLength + i], gpuInputText[s2*iLength + i], flag);
   
                           break;   
                       }
                       else
                       {
                           flag1 = 1;
               //            printf("\nInside inequality else Comparing %c and %c flag %d", gpuInputText[s1*iLength + i], gpuInputText[s2*iLength + i], flag);
                           break;
                       }
                   }
               //    printf("\nAfter Comparing %c and %c flag %d", gpuInputText[s1*iLength + i], gpuInputText[s2*iLength + i], flag);
   
               }

               if(flag1 == 0)
               {
                   flag = 1;
               }

           }
           else if(m < n)
           {
           // printf("\nM < n");

               for(int i = 0; i < m; i++)
               {
             //   printf("\nId %d %c %c", id, gpuInputText[s1*iLength + i], gpuInputText[s2*iLength + i]);

                   if(gpuInputText[s1*iLength + i] <= 122 && 97 <= gpuInputText[s1*iLength + i] )
                       {
                           gpuInputText[s1*iLength + i] = gpuInputText[s1*iLength + i]- 97 + 65;
                       }
                       if(gpuInputText[s2*iLength + i] <= 122 && 97 <= gpuInputText[s2*iLength + i] )
                       {
                           gpuInputText[s2*iLength + i] = gpuInputText[s2*iLength + i]- 97 + 65;
                       }
   
              ///     printf("\nBefore Comparing %c and %c flag %d", gpuInputText[s1*iLength + i], gpuInputText[s2*iLength + i], flag);
                   if(gpuInputText[s1*iLength + i] != gpuInputText[s2*iLength + i])
                   {
                       if(gpuInputText[s1*iLength + i] > gpuInputText[s2*iLength + i])
                       {
                           flag = 1;
              //             printf("\nInside inequality Comparing %c and %c flag %d", gpuInputText[s1*iLength + i], gpuInputText[s2*iLength + i], flag);
   
                           break;   
                       }
                       else
                       {
                //           printf("\nInside inequality else Comparing %c and %c flag %d", gpuInputText[s1*iLength + i], gpuInputText[s2*iLength + i], flag);
                           break;
                       }
                   }
                //   printf("\nAfter Comparing %c and %c flag %d", gpuInputText[s1*iLength + i], gpuInputText[s2*iLength + i], flag);
   
               }
           }
           


			if(flag == 1)			
            {
                //printf("\nSwapping");
				int temp=gpuIndex[id*2 + 1];
				gpuIndex[id*2+1]=gpuIndex[id*2+2];
				gpuIndex[id*2+2]=temp;
                // for(int i =0; i < n; i++)
                //     printf("%d ",gpuIndex[i]);

			}
		}
	}
}



int main(int argc, char **argv)
{
    int n;
    int iLength;
    
	FILE *filePointer;
	char *filename = argv[1]; 
   	filePointer = fopen( filename , "r") ; 
      
    if ( filePointer == NULL ) 
    {
       printf( "input.txt file failed to open." ) ; 
	   return 0;
    }
    
    char newLine;
    fscanf(filePointer, "%d", &n);
    fscanf(filePointer, "%d", &iLength);
    fscanf(filePointer, "%c", &newLine);

    char inputText[(iLength+1)*(n)];
    int length[n];
    char temp;

    //Todo last line may not havenewLine character at end
    for (int i = 0; i < n; i++)
    {
        int temp1 = 0;
        for(int j = 0; j <= iLength; j++)
        {
            temp1++;
            fscanf(filePointer, "%c", &temp);
            if(temp == '\n')
            {
                inputText[i*iLength + j] = '\0';
                break;
            }  
            else
            {
                inputText[i*iLength + j] = temp;
    
            }
        }
        length[i] = temp1;
    }
  
    // cout<<"Length";
    // for(int i = 0; i < n; i++)
    // {
    //     cout<<length[i]<<" ";
    // }
    // cout<<endl;
    // for (size_t i = 0; i < n; i++)
	// {
	//     for (size_t j = 0; j <= iLength; j++)
	//     {
	//     	if(inputText[i*iLength + j] != '\0')
	// 		printf("%c", inputText[i*iLength + j]);
    //         else
    //         {
    //             printf("Null");
    //             break;
    //         }
	//     }
	//     printf("\n");
	// }

    char *gpuInputText;
    hipMalloc(&gpuInputText, (iLength+1)*(n)*sizeof(char));
    hipMemcpy(gpuInputText, inputText, (iLength+1)*(n)*sizeof(char), hipMemcpyHostToDevice);
  
    int *index = (int *)malloc(n*sizeof(int)); 
	for(int i = 0; i < n; i++)
	{
		index[i] = i;
	}	
	
    int *gpuIndex;
    hipMalloc( &gpuIndex, sizeof(int) * (n) );
	hipMemcpy(gpuIndex, index, sizeof(int) * (n), hipMemcpyHostToDevice);

    int *gpuLength;
    hipMalloc( &gpuLength, sizeof(int)*n);
    hipMemcpy(gpuLength, length,  sizeof(int)*n, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    hipEventRecord(start,0);

    for(int i = 0; i < n; i++)
    {
        naiveStringSort <<<ceil(float(n)/BLOCKSIZE) ,BLOCKSIZE >>> (gpuInputText, i, n, gpuIndex, iLength, gpuLength);
    }
    
    hipDeviceSynchronize();

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken by function to execute is: %.6f ms\n", milliseconds);

    hipMemcpy(index, gpuIndex, sizeof(int)*n , hipMemcpyDeviceToHost);


    char *fname = argv[2]; 
    FILE *fptr;
    fptr = fopen(fname,"w");
    
    // cout<<"Final";

    // for(int i=0;i<n;i++)
    // {
    //     cout<<index[i]<<" ";
    // }

    // cout<<endl;


    for (size_t i = 0; i < n; i++)
	{
	    for (size_t j = 0; j <= iLength; j++)
	    {
	    	if(inputText[(index[i])*iLength + j] != '\0')
			    fprintf(fptr, "%c", inputText[(index[i])*iLength + j]);
            else
            {
               // printf("Null");
                break;
            }
	    }
        fprintf(fptr,"\n"); 
    }

       

    return 0;
}