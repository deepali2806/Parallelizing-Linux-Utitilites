#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/iterator/constant_iterator.h>
#include <iostream>
#include <thrust/iterator/zip_iterator.h>
#define BLOCKSIZE 32
//using namespace std;
//Limitation : Tested only for fixed length strings & CAPITAL Letters


__global__ void loadPrefixes(char *gpuInputText, int offset, int iLength, char *gKey1, char *gKey2, int n, int *indexArray)
{
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < n)
    {
        if(offset < iLength && (offset + 1) < iLength )
            {
                gKey1[id] = gpuInputText[(indexArray[id])*iLength + offset];
                gKey2[id] = gpuInputText[(indexArray[id])*iLength + offset + 1];
            }
    }
}

__global__ void findSingleton(int *gSingletonElement, int *gSegId, char *gKey1, char *gKey2, int *gIndexArray,int n, int *gOutputIndex)
{
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id > 0 && id < n-1)
    {
        if(gKey1[id] == gKey1[id-1] && gKey2[id] == gKey2[id-1] && gSegId[id] == gSegId[id-1]) 
        {
            if(gSingletonElement[id] != 1)
            {
                gSingletonElement[id] = 0;
            }
                
            return;
        }
        if(gKey1[id] == gKey1[id+1] && gKey2[id] == gKey2[id+1] && gSegId[id] == gSegId[id+1]) 
        {
            if(gSingletonElement[id] != 1)
            {
                gSingletonElement[id] = 0;
            }
            return;
        }
        else
        {
            gOutputIndex[id] = gIndexArray[id];
            gSingletonElement[id] = 1;
        }
    }
    else if(id == 0)
    {
        if(gKey1[id] != gKey1[id+1] || gKey2[id] != gKey2[id+1] )
        {
            gOutputIndex[id] = gIndexArray[id];
            gSingletonElement[id] = 1;
        }
        else
        {
            if(gSingletonElement[id] != 1)
            {
                gSingletonElement[id] = 0;
            }
                
        }

    }
    else if(id == n-1)
    {
        if(gKey1[id] != gKey1[id-1] || gKey2[id] != gKey2[id-1])
        {
            gOutputIndex[id] = gIndexArray[id];
            gSingletonElement[id] = 1;
        }
        else
        {
            if(gSingletonElement[id] != 1)
            {
                gSingletonElement[id] = 0;
            }
                
        }
    }
}

int main(int argc, char **argv)
{
    int n, iLength;
    FILE *filePointer;
	char *filename = argv[1]; 
   	filePointer = fopen( filename , "r") ; 
      
    if ( filePointer == NULL ) 
    {
       printf( "input.txt file failed to open." ) ; 
	   return 0;
    }

    char newLine;
    fscanf(filePointer, "%d", &n);
    fscanf(filePointer, "%d", &iLength);
    fscanf(filePointer, "%c", &newLine);

    char inputText[iLength*n + n];

    for (int i = 0; i < n; i++)
    {
        for(int j = 0; j <= iLength; j++)
        {
            fscanf(filePointer, "%c", &inputText[i*iLength + j]);
        }
    }

    // for (size_t i = 0; i < n; i++)
    // {
    //     for (size_t j = 0; j < iLength; j++)
    //     {
    //         printf("%c", inputText[i*iLength + j]);
    //     }
    //     printf("\n");
    // }

    thrust::device_vector<int> indexArray(n);
    thrust::sequence (indexArray.begin(), indexArray.end());

    char *gpuInputText;
    hipMalloc(&gpuInputText, iLength*n*sizeof(char));
    hipMemcpy(gpuInputText, inputText, iLength*n*sizeof(char), hipMemcpyHostToDevice);

    //Initial Setup
    thrust::device_vector<int> singletonElement(n, 0);
    thrust::device_vector<int> segId(n, 0);
    thrust::device_vector<char> k1(n);
    thrust::device_vector<char> k2(n);
    thrust::device_vector<int> outputIndex(n, 0);
    
    char *hk1 = (char *)malloc(n*sizeof(char)); 
    char *hk2 = (char *)malloc(n*sizeof(char)); 
    int *hSingletonElement = (int *)malloc(n*sizeof(int)); 
    int *hSegId = (int *)malloc(n*sizeof(int)); 
    int *hOutputIndex = (int *)malloc(n*sizeof(int));
    //Load prefix
    //Check offset < iLength
    int offset = 0;

        int *gSingletonElement = thrust::raw_pointer_cast(&singletonElement[0]);
        char *gKey1 = thrust::raw_pointer_cast(&k1[0]);
        char *gKey2 = thrust::raw_pointer_cast(&k2[0]);
        int *gIndexArray = thrust::raw_pointer_cast(&indexArray[0]);
        int *gSegId = thrust::raw_pointer_cast(&segId[0]);
        int *gOutputIndex = thrust::raw_pointer_cast(&outputIndex[0]);


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    hipEventRecord(start,0);


    // for (int i = 0; i < n; i++)
    // {
    //   k1[i] = gpuInputText[i*iLength + 0];
    //   k2[i] = gpuInputText[i*iLength + 1];
    // }
    
    loadPrefixes<<<ceil(float(n)/BLOCKSIZE) ,BLOCKSIZE>>>(gpuInputText, offset, iLength, gKey1, gKey2, n, gIndexArray);
    hipDeviceSynchronize();
    int numberOfSingletonElements = 0;
    int itr = 0;

    do
    {
        itr++;
        thrust::sort_by_key(thrust::make_zip_iterator(thrust::make_tuple(segId.begin(), k1.begin(), k2.begin())), 
                            thrust::make_zip_iterator(thrust::make_tuple(segId.end(), k1.end(), k2.end())), 
                            indexArray.begin());
        
        // std::cout<<"After Sorting"<<std::endl;
        // for(int i = 0; i < n; i++)
        // {
        //     std::cout<<segId[i]<<" "<< k1[i]<<" "<<k2[i]<<" "<<indexArray[i]<<" "<<std::endl;
        // }

        findSingleton <<<ceil(float(n)/BLOCKSIZE) ,BLOCKSIZE>>>(gSingletonElement, gSegId, gKey1, gKey2, gIndexArray, n, gOutputIndex);
        hipDeviceSynchronize();

       // std::cout<<"Flag"<<std::endl;

        numberOfSingletonElements = 0;

        numberOfSingletonElements = thrust::count(thrust::device, singletonElement.begin(),singletonElement.end(), 1);
       
        // for(int i = 0; i < n; i++)
        // {
        //     if(singletonElement[i] == 1)
        //     {
        //        // numberOfSingletonElements++;
        //     }

        //     std::cout<<singletonElement[i]<<"  ";
        // }
        // std::cout<<""<<std::endl;

        // std::cout<<"OutpUt array"<<std::endl;

        // for(int i = 0; i < n; i++)
        // {
        //     std::cout<<outputIndex[i]<<"  ";
        // }
        // std::cout<<""<<std::endl;

        //Generate Segment IDs sequentially
        int cnt = 0;

        hipMemcpy(hk1, gKey1, sizeof(char)*n , hipMemcpyDeviceToHost);
        hipMemcpy(hk2, gKey2, sizeof(char)*n , hipMemcpyDeviceToHost);
        hipMemcpy(hSingletonElement, gSingletonElement, sizeof(int)*n , hipMemcpyDeviceToHost);
        hipMemcpy(hSegId, gSegId, sizeof(int)*n , hipMemcpyDeviceToHost);

        // segId[0] = cnt;
        // for(int i = 1; i < n; i++)
        // {
        //     if(singletonElement[i] == 1)
        //     {
        //         cnt++;
        //         segId[i] = cnt;
        //     }
        //     else if(singletonElement[i] == 0 && (k1[i] == k1[i-1] && k2[i] == k2[i-1]))
        //     {
        //         segId[i] = segId[i-1]; 
        //     }
        //     else if(singletonElement[i] == 0 && (k1[i] != k1[i-1] || k2[i] != k2[i-1]))
        //     {
        //         cnt++;
        //         segId[i] = cnt;
        //     }   
        // }

        hSegId[0] = cnt;
        for(int i = 1; i < n; i++)
        {
            if(hSingletonElement[i] == 1)
            {
                cnt++;
                hSegId[i] = cnt;
            }
            else if(hSingletonElement[i] == 0 && (hk1[i] == hk1[i-1] && hk2[i] == hk2[i-1]))
            {
                hSegId[i] = hSegId[i-1]; 
            }
            else if(hSingletonElement[i] == 0 && (hk1[i] != hk1[i-1] || hk2[i] != hk2[i-1]))
            {
                cnt++;
                hSegId[i] = cnt;
            }   
        }
        hipMemcpy(gSegId, hSegId, sizeof(int)*n , hipMemcpyHostToDevice);

        // std::cout<<"Segments"<<std::endl;

        // for(int i = 0; i < n; i++)
        // {
        //     std::cout<<segId[i]<<"  ";
        // }
        // std::cout<<""<<std::endl;

        offset = offset + 2;
       
        //Load Prefixes
        // for (int i = 0; i < n; i++)
        // {
        //     if(offset < iLength && (offset + 1) < iLength )
        //     {
        //         k1[i] = gpuInputText[i*iLength + offset];
        //         k2[i] = gpuInputText[i*iLength + offset + 1];
        //     }
          
        // }
        loadPrefixes<<<ceil(float(n)/BLOCKSIZE) ,BLOCKSIZE>>>(gpuInputText, offset, iLength, gKey1, gKey2, n, gIndexArray);
        hipDeviceSynchronize();

        // std::cout<<"Prefix Loaded"<<std::endl;

        // for(int i = 0; i < n; i++)
        // {
        //     std::cout<<segId[i]<<" "<< k1[i]<<" "<<k2[i]<<" "<<indexArray[i]<<" "<<std::endl;
        // }
        // std::cout<<"Iteration "<<itr<<std::endl;
        // std::cout<<"Nuber of singlton Elements "<<numberOfSingletonElements<<std::endl;
        // std::cout<<"Offset "<<offset<<std::endl;

    }while( numberOfSingletonElements != n);
    
    hipDeviceSynchronize();

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken by function to execute is: %.6f ms\n", milliseconds);


        // std::cout<<"After All iteration OutpUt array"<<std::endl;

        // for(int i = 0; i < n; i++)
        // {
        //     std::cout<<outputIndex[i]<<"  ";
        // }
        // std::cout<<""<<std::endl;

        // std::cout<<"Final OutpUt"<<std::endl;
        hipMemcpy(hOutputIndex, gOutputIndex, sizeof(int)*n , hipMemcpyDeviceToHost);

        char *fname = argv[2]; 
        FILE *fptr;
        fptr = fopen(fname,"w");

        for(int i = 0; i < n; i++)
        {
            for(int j = 0; j < iLength; j++)
            {
                fprintf(fptr, "%c", inputText[hOutputIndex[i]*iLength + j]);
               // std::cout<< inputText[outputIndex[i]*iLength + j]<<"";
            }
            fprintf(fptr,"\n"); 
        }
        fprintf(fptr,"\n"); 

    return 0;
}

